#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <iostream>
#include <string>
#include <algorithm>
#include <nixl_descriptors.h>
#include <nixl_params.h>
#include <nixl.h>
#include <cassert>
#include "stream/metadata_stream.h"
#include "serdes/serdes.h"

#define CUDA_THREADS 512
#define NUM_STREAMS 8
#define TRANSFER_NUM_BUFFER 32
#define TRANSFER_NUM 1
#define SIZE 1024
#define INITIATOR_VALUE 0xbb
#define VOLATILE(x) (*(volatile typeof(x) *)&(x))
#define INITIATOR_THRESHOLD_NS 50000 //50us
#define USE_NVTX 1

#if USE_NVTX
#include <nvtx3/nvToolsExt.h>

const uint32_t colors[] = { 0xff00ff00, 0xff0000ff, 0xffffff00, 0xffff00ff, 0xff00ffff, 0xffff0000, 0xffffffff };
const int num_colors = sizeof(colors)/sizeof(uint32_t);

#define DEVICE_GET_TIME(globaltimer) asm volatile("mov.u64 %0, %globaltimer;" : "=l"(globaltimer))

#define PUSH_RANGE(name,cid) { \
	int color_id = cid; \
	color_id = color_id%num_colors;\
	nvtxEventAttributes_t eventAttrib = {0}; \
	eventAttrib.version = NVTX_VERSION; \
	eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE; \
	eventAttrib.colorType = NVTX_COLOR_ARGB; \
	eventAttrib.color = colors[color_id]; \
	eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII; \
	eventAttrib.message.ascii = name; \
	nvtxRangePushEx(&eventAttrib); \
}
#define POP_RANGE nvtxRangePop();
#else
#define PUSH_RANGE(name,cid)
#define POP_RANGE
#endif

static void checkCudaError(hipError_t result, const char *message) {
	if (result != hipSuccess) {
		std::cerr << message << " (Error code: " << result << " - "
				   << hipGetErrorString(result) << ")" << std::endl;
		exit(EXIT_FAILURE);
	}
}

__global__ void target_kernel(uintptr_t addr)
{
	uint8_t ok = 1;
	uintptr_t buffer_addr = addr + (threadIdx.x * SIZE);

	printf(">>>>>>> CUDA target waiting on buffer %d addr %lx size %d\n",
			threadIdx.x, buffer_addr, (uint32_t)SIZE);

	while(VOLATILE(((uint8_t*)buffer_addr)[0]) == 0);

	for (int i = 0; i < (int)SIZE; i++) {
		if (((uint8_t*)buffer_addr)[i] != INITIATOR_VALUE) {
			printf(">>>>>>> CUDA target byte %x is wrong\n", i);
			ok = 1;
		}
	}
	if (ok == 1)
		printf(">>>>>>> CUDA target, all bytes received!\n");
	else
		printf(">>>>>>> CUDA target, not all received bytes are ok!\n");
}

int launch_target_wait_kernel(hipStream_t stream, uintptr_t addr)
{
	hipError_t result = hipSuccess;

	/* Check no previous CUDA errors */
	result = hipGetLastError();
	if (result != hipSuccess) {
		fprintf(stderr, "[%s:%d] cuda failed with %s", __FILE__, __LINE__, hipGetErrorString(result));
		return -1;
	}

	target_kernel<<<1, TRANSFER_NUM_BUFFER, 0, stream>>>(addr);
	result = hipGetLastError();
	if (result != hipSuccess) {
		fprintf(stderr, "[%s:%d] cuda failed with %s", __FILE__, __LINE__, hipGetErrorString(result));
		return -1;
	}

	return 0;
}

__global__ void initiator_kernel(uintptr_t addr)
{
	unsigned long long start, end;
	// Each block updates a buffer in this transfer
	uintptr_t block_address = (addr + (blockIdx.x * SIZE));

	/* Simulate a longer CUDA kernel to process initiator data */
	DEVICE_GET_TIME(start);

	for (int i = threadIdx.x; i < SIZE; i+=blockDim.x)
		((uint8_t*)block_address)[i] = INITIATOR_VALUE;

	__syncthreads();

	do {
		DEVICE_GET_TIME(end);
	} while (end - start < INITIATOR_THRESHOLD_NS);
}

int launch_initiator_send_kernel(hipStream_t stream, uintptr_t addr)
{
	hipError_t result = hipSuccess;

	/* Check no previous CUDA errors */
	result = hipGetLastError();
	if (result != hipSuccess) {
		fprintf(stderr, "[%s:%d] cuda failed with %s", __FILE__, __LINE__, hipGetErrorString(result));
		return -1;
	}

	// Block = # buffers x transfer
	initiator_kernel<<<TRANSFER_NUM_BUFFER, CUDA_THREADS, 0, stream>>>(addr);
	result = hipGetLastError();
	if (result != hipSuccess) {
		fprintf(stderr, "[%s:%d] cuda failed with %s", __FILE__, __LINE__, hipGetErrorString(result));
		return -1;
	}

	return 0;
}

/**
 * This test does p2p from using PUT.
 * intitator -> target so the metadata and
 * desc list needs to move from
 * target to initiator
 */

bool allBytesAre(void* buffer, size_t size, uint8_t value) {
	uint8_t* byte_buffer = static_cast<uint8_t*>(buffer); // Cast void* to uint8_t*
	// Iterate over each byte in the buffer
	for (size_t i = 0; i < size; ++i) {
		if (byte_buffer[i] != value) {
			return false; // Return false if any byte doesn't match the value
		}
	}
	return true; // All bytes match the value
}

std::string recvFromTarget(int port) {
	nixlMDStreamListener listener(port);
	listener.setupListenerSync();
	listener.acceptClient();
	return listener.recvFromClient();
}

void sendToInitiator(const char *ip, int port, std::string data) {
	nixlMDStreamClient client(ip, port);
	client.connectListenerSync();
	client.sendData(data);
}

int main(int argc, char *argv[]) {
	int                     peer_port;
	nixl_status_t           ret = NIXL_SUCCESS;
	uint8_t                 *data_address;
	std::string             role;
	std::string             processing;
	const char              *peer_ip;
	nixl_blob_t             remote_desc;
	nixl_blob_t             metadata;
	nixl_blob_t             remote_metadata;
	int                     status = 0;

	/** NIXL declarations */
	/** Agent and backend creation parameters */
	nixlAgentConfig cfg(true);
	nixl_b_params_t params;
	nixlBlobDesc    buf[TRANSFER_NUM_BUFFER];
	nixlBackendH    *doca;
	hipStream_t    stream[NUM_STREAMS];
	/** Serialization/Deserialization object to create a blob */
	nixlSerDes *serdes        = new nixlSerDes();
	nixlSerDes *remote_serdes = new nixlSerDes();
	std::string target_name;

	/** Descriptors and Transfer Request */
	nixl_reg_dlist_t  dram_for_doca(DRAM_SEG);
	nixlXferReqH      *treq[TRANSFER_NUM];

	/** Argument Parsing */
	if (argc < 5) {
		std::cout <<"Enter the required arguments\n" << std::endl;
		std::cout <<"<Role> <Peer IP> <Peer Port> <CPU or GPU processing>"
				  << std::endl;
		exit(-1);
	}

	role = std::string(argv[1]);
	std::transform(role.begin(), role.end(), role.begin(), ::tolower);
	if (!role.compare("initiator") && !role.compare("target")) {
			std::cerr << "Invalid role. Use 'initiator' or 'target'."
					  << "Currently "<< role <<std::endl;
			return 1;
	}

	peer_ip   = argv[2];
	peer_port = std::stoi(argv[3]);
	processing = std::string(argv[4]);
	std::transform(processing.begin(), processing.end(), processing.begin(), ::tolower);
	if (!processing.compare("cpu") && !processing.compare("gpu")) {
			std::cerr << "Invalid type of processing. Use 'cpu' or 'gpu'."
					  << "Currently "<< processing <<std::endl;
			return 1;
	}

	/*** End - Argument Parsing */

	checkCudaError(hipSetDevice(0), "Failed to set device");
	hipFree(0);

	/** Common to both Initiator and Target */
	std::cout << "Starting Agent for "<< role << "\n";
	nixlAgent     agent(role, cfg);
	params["network_devices"] = "mlx5_0";
	params["gpu_devices"] = "0";
	PUSH_RANGE("createBackend", 0)
	agent.createBackend("DOCA", params, doca);
	POP_RANGE

	nixl_opt_args_t extra_params;
	extra_params.backends.push_back(doca);

	checkCudaError(hipMalloc(&data_address, SIZE * TRANSFER_NUM_BUFFER), "Failed to allocate CUDA buffer 0");
	checkCudaError(hipMemset((void*)data_address, 0, SIZE * TRANSFER_NUM_BUFFER), "Failed to memset CUDA buffer 0");

	if (role != "target") {
		std::cout << "Allocating for initiator : "
				  << TRANSFER_NUM_BUFFER << " buffers "
				  << SIZE << " Bytes each "
				  << (void*)data_address << " address "
				  << std::endl;
	} else {
		std::cout << "Allocating for target : "
				  << TRANSFER_NUM_BUFFER << " buffers "
				  << SIZE << " Bytes each "
				  << (void*)data_address << " address "
				  << std::endl;
	}

	for (int i = 0; i < TRANSFER_NUM_BUFFER; i++) {
		buf[i].addr  = (uintptr_t)(data_address + (i * SIZE));
		buf[i].len   = SIZE;
		buf[i].devId = 0;
		dram_for_doca.addDesc(buf[i]);
	}
	/** Register memory in both initiator and target */
	agent.registerMem(dram_for_doca, &extra_params);
	agent.getLocalMD(metadata);

	std::cout << " Start Control Path metadata exchanges \n";
	if (role == "target") {
		nixlMDStreamClient client(peer_ip, peer_port);
		client.connectListenerSync();

		std::cout << " Desc List from Target to Initiator\n";
		dram_for_doca.print();

		/** Sending both metadata strings together */
		assert(serdes->addStr("AgentMD", metadata) == NIXL_SUCCESS);
		assert(dram_for_doca.trim().serialize(serdes) == NIXL_SUCCESS);

		std::cout << " Serialize Metadata to string and Send to Initiator\n";
		std::cout << " \t -- To be handled by runtime - currently sent via a TCP Stream\n";
		// sendToInitiator(peer_ip, peer_port, serdes->exportStr());
		client.sendData(serdes->exportStr());
		std::cout << " End Control Path metadata exchanges \n";

		std::cout << " Start Data Path Exchanges \n";
		std::cout << " Waiting to receive Data from Initiator\n";

		checkCudaError(hipStreamCreateWithFlags(&stream[0], hipStreamNonBlocking), "Failed to create CUDA stream");

		/* 1 target CUDA kernel per transfer. Each thread will check a single buffer in the transfer */
		for (int i = 0; i < TRANSFER_NUM; i++) {
			printf("Launching kernel %d addr %lx\n", i, (uintptr_t)(data_address));
			launch_target_wait_kernel(stream[0], (uintptr_t)(data_address));
			// while(1) {
			// 	nixl_notifs_t notifs;
			// 	nixl_status_t ret = agent.getNotifs(notifs);
			// }
			
			hipStreamSynchronize(stream[0]);
		}

		std::cout << " DOCA Transfer completed!\n";

		hipStreamDestroy(stream[0]);
	} else {
		std::cout << " Receive metadata from Target \n";
		std::cout << " \t -- To be handled by runtime - currently received via a TCP Stream\n";

		nixlMDStreamListener listener(peer_port);
		listener.setupListenerSync();
		listener.acceptClient();

		// nixl_opt_args_t md_extra_params;
		// md_extra_params.ipAddr = peer_ip;
		// md_extra_params.port = peer_port;
		// agent.fetchRemoteMD(target_name, &md_extra_params);
		// agent.sendLocalMD(&md_extra_params);
		std::string rrstr = listener.recvFromClient(); //recvFromTarget(peer_port);
		remote_serdes->importStr(rrstr);
		remote_metadata = remote_serdes->getStr("AgentMD");
		assert (remote_metadata != "");
		agent.loadRemoteMD(remote_metadata, target_name);

		std::cout << " Verify Deserialized Target's Desc List at Initiator\n";
		nixl_xfer_dlist_t dram_target_doca(remote_serdes);
		nixl_xfer_dlist_t dram_initiator_doca = dram_for_doca.trim();
		dram_target_doca.print();
		std::cout << " Got metadata from " << target_name << " \n";
		std::cout << " Create transfer request with DOCA backend\n ";

		PUSH_RANGE("createXferReq", 1)

		if (processing.compare("gpu") == 0)
	        for (int i = 0; i < NUM_STREAMS; i++)
    	        checkCudaError(hipStreamCreateWithFlags(&stream[i], hipStreamNonBlocking),
								"Failed to create CUDA stream");

		//Multiple transfer requests, same buffers
		for (int transfer_idx = 0; transfer_idx < TRANSFER_NUM; transfer_idx++) {
			if (processing.compare("gpu") == 0) {
				extra_params.customParam.resize(sizeof(uintptr_t));
				*((uintptr_t*) extra_params.customParam.data()) = (uintptr_t)stream[transfer_idx];
			}
			extra_params.notifMsg = "test";
			extra_params.hasNotif = true;
			ret = agent.createXferReq(NIXL_WRITE, dram_initiator_doca, dram_target_doca,
							"target", treq[transfer_idx], &extra_params);
			if (ret != NIXL_SUCCESS) {
				std::cerr << "Error creating transfer request\n";
				exit(-1);
			}
		}
		POP_RANGE

		std::cout << "Launch initiator send kernel on stream\n";

		/* Synthetic simulation of GPU processing data before sending */
		if (processing.compare("gpu") == 0) {
			for (int transfer_idx = 0; transfer_idx < TRANSFER_NUM; transfer_idx++) {
				std::cout << " Prepare data, GPU mode, transfer " << transfer_idx << " \n ";
				PUSH_RANGE("InitData", 2)
				launch_initiator_send_kernel(stream[transfer_idx], (uintptr_t)(data_address));
				POP_RANGE

				std::cout << " Post the request with DOCA backend transfer " << transfer_idx << " \n ";
				PUSH_RANGE("postXferReq", 3)
				status = agent.postXferReq(treq[transfer_idx]);
				assert(status >= NIXL_SUCCESS);
				POP_RANGE
			} 
		} else {
			/* Synthetic simulation of CPU processing data before sending */
			for (int transfer_idx = 0; transfer_idx < TRANSFER_NUM; transfer_idx++) {
				std::cout << " Prepare data, CPU mode, transfer " << transfer_idx << " \n ";
				PUSH_RANGE("InitData", 2)
				hipMemset((void*)data_address, INITIATOR_VALUE, TRANSFER_NUM_BUFFER * SIZE);
				POP_RANGE

				std::cout << " Post the request with DOCA backend transfer " << transfer_idx << " \n ";
				PUSH_RANGE("postXferReq", 3)
				status = agent.postXferReq(treq[transfer_idx]);
				assert(status >= NIXL_SUCCESS);
				POP_RANGE
			}
		}

		std::cout << " Initiator posted Data Path transfer\n";
		std::cout << " Waiting for completion\n";

		PUSH_RANGE("getXferStatus", 4)
		for (int transfer_idx = 0; transfer_idx < TRANSFER_NUM; transfer_idx++) {
			while (status != NIXL_SUCCESS) {
				status = agent.getXferStatus(treq[transfer_idx]);
				assert(status >= NIXL_SUCCESS);
			}
		}
		POP_RANGE
		std::cout << " Completed Sending " << TRANSFER_NUM << " transfers using DOCA backend\n";
		for (int transfer_idx = 0; transfer_idx < TRANSFER_NUM; transfer_idx++)
			agent.releaseXferReq(treq[transfer_idx]);
	
		if (processing.compare("gpu") == 0) {
			for (int transfer_idx = 0; transfer_idx < TRANSFER_NUM; transfer_idx++) {
				hipStreamSynchronize(stream[transfer_idx]);
				hipStreamDestroy(stream[transfer_idx]);
			}
		}
	}

	std::cout <<"Cleanup.. \n";
	
	agent.deregisterMem(dram_for_doca, &extra_params);
	// hipFree(data_address);

	if (role == "target")
		delete serdes;
	else
		delete remote_serdes;

	std::cout <<"Exit.. \n";

	return 0;
}
